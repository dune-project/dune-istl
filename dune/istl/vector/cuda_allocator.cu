#include <hip/hip_runtime.h>
#include <dune/istl/vector/cuda_allocator.hh>

using namespace Dune;
using namespace Dune::Memory;

template <typename DT_>
typename std::allocator<DT_>::pointer CudaAllocator<DT_>::allocate(size_t n, typename std::allocator<void>::const_pointer /*hint*/)
{
  void * r;
  hipError_t status = hipMalloc(&r, n * sizeof(DT_));
  if (status != hipSuccess)
    throw new std::bad_alloc;

  return (DT_*)r;
}

template <typename DT_>
void CudaAllocator<DT_>::deallocate(typename std::allocator<DT_>::pointer p, size_t /*n*/)
{
  hipFree((void*) p);
}

template typename std::allocator<float>::pointer CudaAllocator<float>::allocate(size_t n, typename std::allocator<void>::const_pointer);
template typename std::allocator<double>::pointer CudaAllocator<double>::allocate(size_t n, typename std::allocator<void>::const_pointer);
template void CudaAllocator<float>::deallocate(typename std::allocator<float>::pointer, size_t);
template void CudaAllocator<double>::deallocate(typename std::allocator<double>::pointer, size_t);
