#include <hip/hip_runtime.h>
#include <dune/istl/vector/cuda_backend.hh>

using namespace Dune;
using namespace Dune::Cuda;

template <typename DT_>
void Dune::Cuda::upload(DT_ * dst, const DT_ * src, size_t count)
{
  hipMemcpy(dst, src, count * sizeof(DT_), hipMemcpyHostToDevice);
}

template <typename DT_>
void Dune::Cuda::download(DT_ * dst, const DT_ * src, size_t count)
{
  hipMemcpy(dst, src, count * sizeof(DT_), hipMemcpyDeviceToHost);
}

template <typename DT_>
void Dune::Cuda::copy(DT_ * dst, const DT_ * src, size_t count)
{
  hipMemcpy(dst, src, count * sizeof(DT_), hipMemcpyDeviceToDevice);
}

template <typename DT_>
void Dune::Cuda::set(DT_ * dst, const DT_ & val)
{
  hipMemcpy(dst, &val, sizeof(DT_), hipMemcpyHostToDevice);
}

template <typename DT_>
DT_ Dune::Cuda::get(DT_ * src)
{
  DT_ result;
  hipMemcpy(&result, src, sizeof(DT_), hipMemcpyDeviceToHost);
  return result;
}

template void Dune::Cuda::upload(float *, const float *, size_t);
template void Dune::Cuda::upload(double *, const double *, size_t);
template void Dune::Cuda::download(float *, const float *, size_t);
template void Dune::Cuda::download(double *, const double *, size_t);
template void Dune::Cuda::copy(float *, const float *, size_t);
template void Dune::Cuda::copy(double *, const double *, size_t);
template void Dune::Cuda::set(float *, const float &);
template void Dune::Cuda::set(double *, const double &);
template float Dune::Cuda::get(float *);
template double Dune::Cuda::get(double *);
